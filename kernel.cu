#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


const int N = 16;
const int a[N] = {10,1,8,-1,0,-2,3,5,-2,-3,2,7,0,11,0,2};
const int b[N] = {10,1,8,-1,0,-2,3,5,-2,-3,2,7,0,11,0,2};


__global__ void reduce1(int *a)
{
	int tid = threadIdx.x;
	for (int i = 1; i < blockDim.x; i *= 2)
	{
		if (tid % (2 * i) == 0)
			a[tid] += a[tid + i];
	}
}

__global__ void reduce2(int *a)
{
	int tid = threadIdx.x;
	for (int i = 1; i < blockDim.x; i *= 2)
	{
		int idx = 2 * i * tid;

		if (idx < blockDim.x)
			a[idx] += a[idx + i];
	}
}

__global__ void reduce3(int *a)
{
	int tid = threadIdx.x;
	for (unsigned int i = blockDim.x / 2; i > 0; i = i / 2)
	{

		if (tid < i)
			a[tid] += a[tid + i];
	}

}

__global__ void dot(int *a, int *b)
{
	int tid = threadIdx.x;
	a[tid] *= b[tid];
	for (int i = 1; i < blockDim.x; i *= 2)
	{
		int idx = 2 * i * tid;

		if (idx < blockDim.x)
			a[idx] += a[idx + i];
	}
}

int main()
{
    int c[N] = { 0 };
	int *dev_a = 0;
	int *dev_b = 0;

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	dot <<<1, N>>>(dev_a, dev_b);
	hipMemcpy(c, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
	printf("%d\n", c[0]);//395 expected

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	reduce1 << <1, N >>>(dev_a);
	hipMemcpy(c, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
	printf("%d\n", c[0]);//41 expected

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	reduce2 << <1, N >>>(dev_a);
	hipMemcpy(c, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
	printf("%d\n", c[0]);//41 expected

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	reduce3 << <1, N >>>(dev_a);
	hipMemcpy(c, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
	printf("%d\n", c[0]); //41 expected


    return 0;
}
