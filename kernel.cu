#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


const int N = 512;
//const float a[N] = {10,1,8,-1,0,-2,3,5,-2,-3,2,7,0,11,0,2};
//const float b[N] = {10,1,8,-1,0,-2,3,5,-2,-3,2,7,0,11,0,2};
//const float xx[N] = { -4,-3.5,-3,-2.5,-2,-1.5,-1,-0.5,0,0.5,1,1.5,2,2.5,3,3.5 };


__global__ void reduce1(int *a)
{
	int tid = threadIdx.x;
	for (int i = 1; i < blockDim.x; i *= 2)
	{
		if (tid % (2 * i) == 0)
			a[tid] += a[tid + i];
	}
}

__global__ void reduce2(int *a)
{
	int tid = threadIdx.x;
	for (int i = 1; i < blockDim.x; i *= 2)
	{
		int idx = 2 * i * tid;

		if (idx < blockDim.x)
			a[idx] += a[idx + i];
	}
}

__global__ void reduce3(float *x)
{
	int tid = threadIdx.x;

	x[tid] = x[tid] * x[tid] * (x[tid + 1] - x[tid]);
	x[N - 1] = 0;

	for (unsigned int i = blockDim.x / 2; i > 0; i = i / 2)
	{

		if (tid < i)
			x[tid] += x[tid + i];
	}

}

float*  integration(float low, float high) 
{
	float x[N] = {0};
	float delta = (high - low) / (N - 1);

	for (size_t i = 0; i < N; i++)
	{
		x[i] = low + i * delta;
		/*printf("%.1f\n", x[i]);*/
	}

	float c[N] = { 0 }; 
	float *dev_a = 0;
	hipMalloc((void**)&dev_a, N * sizeof(float));
	hipMemcpy(dev_a, x, N * sizeof(float), hipMemcpyHostToDevice);
	reduce3 << <1, N >> >(dev_a);
	hipMemcpy(c, dev_a, N * sizeof(float), hipMemcpyDeviceToHost);

	printf("%.1f\n", c[0]);


	return x;

}


__global__ void dot(int *a, int *b)
{
	int tid = threadIdx.x;
	a[tid] *= b[tid];
	for (int i = 1; i < blockDim.x; i *= 2)
	{
		int idx = 2 * i * tid;

		if (idx < blockDim.x)
			a[idx] += a[idx + i];
	}
}

__global__ void reduce(int* input, int* output) 
{
	__shared__ int* data;

	int tid = threadIdx.x;
	data[tid] = input[tid];

	__syncthreads();

	for (int i = blockDim.x / 2; i > 0; i = i / 2)
	{
		if (tid < i)
		{
			data[tid] += data[tid + i];
		}
		__syncthreads();
	}

	if (tid == 0) output[blockIdx.x] = data[0];
}

int main()
{
	//float c[N] = { 0 };
	//float *dev_a = 0;
	//float *dev_b = 0;

	//hipMalloc((void**)&dev_a, N * sizeof(float));
	//hipMalloc((void**)&dev_b, N * sizeof(float));

	//hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);
	//dot <<<1, N>>>(dev_a, dev_b);
	//hipMemcpy(c, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
	//printf("%d\n", c[0]);//395 expected

	//hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	//reduce1 <<<1, N >>>(dev_a);
	//hipMemcpy(c, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
	//printf("%d\n", c[0]);//41 expected

	//hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	//reduce2 <<<1, N >>>(dev_a);
	//hipMemcpy(c, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
	//printf("%d\n", c[0]);//41 expected


	integration(-4, 4);

	//hipMemcpy(dev_a, xx, N * sizeof(float), hipMemcpyHostToDevice);
	//reduce3 <<<1, N >>>(dev_a);
	//hipMemcpy(c, dev_a, N * sizeof(float), hipMemcpyDeviceToHost);
	//printf("%.1f\n", c[0]);

	//hipMemcpy(dev_b, xx, N * sizeof(float), hipMemcpyHostToDevice);
	//integration << <1, N >> > (dev_b);
	//hipMemcpy(c, dev_b, N * sizeof(float), hipMemcpyDeviceToHost);

	

	//hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	//reduce <<<2, N / 2>>>(dev_a, dev_b);
	//hipMemcpy(c, dev_b, N * sizeof(int), hipMemcpyDeviceToHost);
	//printf("%d\n", c[0]); //41 expected


    return 0;
}
